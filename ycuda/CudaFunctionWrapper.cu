#include <hip/hip_runtime.h> // need CUDA_VERSION
#include <hipDNN.h>

#include <ycuda/CudaFunctionWrapper.h>

namespace ycuda{

hipError_t CallCudaMemcpy(float* src, float* dst, size_t count, hipMemcpyKind kind)
{
	return hipMemcpy(src, dst, count, kind);
}
hipError_t CallCudaFree(unsigned char* ptr)
{
	return hipFree((void*)ptr);
}
hipError_t CallCudaFree(int* ptr)
{
	return hipFree((void*)ptr);
}
hipError_t CallCudaFree(float* ptr)
{
	return hipFree((void*)ptr);
}
hipError_t CallCudaMallocManaged(float** ptr, size_t size)
{
	return hipMallocManaged((void**)ptr, size);
}
hipError_t CallCudaMallocManaged(int** ptr, size_t size)
{
	return hipMallocManaged((void**)ptr, size);
}
hipError_t CallCudaMallocManaged(unsigned char** ptr, size_t size)
{
	return hipMallocManaged((void**)ptr, size);
}
hipError_t CallCudaDeviceSYnchronize()
{
	return hipDeviceSynchronize();
}

}

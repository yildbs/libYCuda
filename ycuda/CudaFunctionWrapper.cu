#include <hip/hip_runtime.h> // need CUDA_VERSION
#include <hipDNN.h>

#include "CudaFunctionWrapper.h"

namespace ycuda{

hipError_t CallCudaMemcpy(float* src, float* dst, size_t count, hipMemcpyKind kind)
{
	return hipMemcpy(src, dst, count, kind);
}
hipError_t CallCudaFree(float* ptr)
{
	return hipFree((void*)ptr);
}
hipError_t CallCudaMalloc(float** ptr, size_t size)
{
	return hipMalloc(ptr, size);
}
hipError_t CallCudaMallocManaged(float** ptr, size_t size)
{
	return hipMallocManaged((void**)ptr, size);
}
hipError_t CallCudaMallocManaged(unsigned char** ptr, size_t size)
{
	return hipMallocManaged((void**)ptr, size);
}
hipError_t CallCudaDeviceSYnchronize()
{
	return hipDeviceSynchronize();
}

}
